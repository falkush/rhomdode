#include "hip/hip_runtime.h"
﻿#define _USE_MATH_DEFINES

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>

static int nbblocks = 1000;

static uint8_t* buffer = 0;
static double* vecl = 0;
static bool* blocks = 0;
static uint8_t* blockcol = 0;

static double* norm0 = 0;
static double* norm1 = 0;
static double* norm2 = 0;

static double* point0 = 0;
static double* point1 = 0;
static double* point2 = 0;

static double* polp0 = 0;
static double* polp1 = 0;
static double* polp2 = 0;

static double* v0p0 = 0;
static double* v0p1 = 0;
static double* v0p2 = 0;

static double* v1p0 = 0;
static double* v1p1 = 0;
static double* v1p2 = 0;

static int* mir = 0;

static uint8_t* stars = 0;

static int skyr = 96;
static int skyg = 149;
static int skyb = 217;

__device__ double skyfunc(double a, double b, double c, double d, double e, double f, double x)
{
	return e*x+(a*(2.0*b*x+c)*sqrt(x*(b*x+c)+d))/(4.0*b)-(a*(c*c-4.0*b*d)*log(2.0*sqrt(b)*sqrt(x*(b*x+c)+d)+2.0*b*x+c))/(8.0*sqrt(b)*sqrt(b)*sqrt(b))+f*b*x*x*x/3.0+f*c*x*x/2+f*d*x;
}

__device__ double gauss(double x)
{
	double ret;
	double sigma = 0.027;

	ret = 1.0 / (sigma * sqrt(2.0 * M_PI));
	ret *= exp((-1.0 / 2.0) * x * x * (1.0 / sigma) * (1.0 / sigma));

	return ret;
}

__device__ int rnbw(int nbframe)
{
	int r=0, g=0, b=0;
	double tmp;
	double x;
	
		x = fmod(nbframe * 0.006, 1.0);
		tmp = fmod(x, 1.0 / 6.0);

		if (x < 1.0/6.0)
		{
			r = 255;
			g = 1530 * tmp;
		}
		else if (x < 1.0/3.0)
		{
			g = 255;
			r = 255 - 1530 * tmp;
		}
		else if (x < 0.5)
		{
			g = 255;
			b = 1530 * tmp;
		}
		else if (x < 2.0 / 3.0)
		{
			b = 255;
			g =255 - 1530 * tmp;
		}
		else if (x < 5.0 / 6.0)
		{
			b = 255;
			r = 1530 * tmp;
		}
		else
		{
			r = 255;
			b = 255 - 1530 * tmp;
		}

		return r+256*g+256*256*b;
}

__global__ void remblock(bool* blocks,int remidx)
{
	blocks[remidx] = false;
}

__global__ void addblock(bool* blocks, int addidx)
{
	blocks[addidx] = true;
}

__global__ void changecol (uint8_t* blockcol, int buildidx, uint8_t col)
{
	blockcol[buildidx] = col;
}

__global__ void setstars(uint8_t* stars)
{
	int i;
	int tmp = blockIdx.x * blockDim.x + threadIdx.x;

	int rand = tmp;

	for (i = 0; i < 10; i++) rand = (60493 *rand+11)% 115249;

	if ((rand)%200==0)
	{
		stars[tmp] = 255*rand/ 115249;
	}
	else
	{
		stars[tmp] = 0;
	}
}

__global__ void setplanet(bool* blocks, uint8_t* blockcol)
{
	int tmp, i, j, k,l;
	int rand;


	tmp = blockIdx.x * blockDim.x + threadIdx.x;
	rand = tmp;
	i = tmp % 500;
	tmp -= i;
	tmp /= 500;
	j = tmp%1000;
	tmp -= j;
	k = tmp / 1000;

	int blockidx;
	int nbblocks = 1000;
	double disttmp;
	int tmp2;
	int tmp3;

	for (l = 0; l < 100; l++) rand = (60493 * rand + 11) % 479001599;

				if ((j + k) % 2 == 0) tmp2 = 2 * i;
				else tmp2 = 2 * i + 1;
				
				disttmp = sqrt((tmp2 - 500 + 0.5) * (tmp2 - 500 + 0.5) + (j - 500 + 0.5) * (j - 500 + 0.5) + (k - 500 + 0.5) * (k - 500 + 0.5));
				blockidx = i + nbblocks * j + nbblocks * nbblocks * k;



				if (disttmp < 32)
				{
					blocks[blockidx] = true;
					blockcol[blockidx] = 255;
				}
				else if (disttmp < 64)
				{
					blocks[blockidx] = true;
					blockcol[blockidx] = 215;
				}
				else if (disttmp < 120)
				{
					blocks[blockidx] = true;
					if(rand%8==0) blockcol[blockidx] = 65;
					else if(rand%8==1 || rand%8==2) blockcol[blockidx] = 29;
					else blockcol[blockidx] = 35;
				}
				else if (disttmp < 180)
				{
					blocks[blockidx] = true;
					if (rand % 6 == 0) blockcol[blockidx] = 9;
					else if (rand % 6 == 1) blockcol[blockidx] = 11;
					else if (rand % 6 == 2) blockcol[blockidx] = 16;
					else blockcol[blockidx] = 17;
				}
				else if (disttmp < 250)
				{
					blocks[blockidx] = true;
					if (rand % 6 == 0) blockcol[blockidx] = 4;
					else if (rand % 6 == 1) blockcol[blockidx] = 3;
					else blockcol[blockidx] = 5;
				}
				else if (disttmp < 497)
				{
					tmp3 = disttmp - 375;
					if (tmp3 < 0) tmp3 *= -1;
					if(tmp3>15) blocks[blockidx] = true;

					if (rand % 10 == 0) blockcol[blockidx] = 43;
					else if (rand % 10 == 1) blockcol[blockidx] = 86;
					else if (rand % 10 == 2) blockcol[blockidx] = 129;
					else if (rand % 10 == 3) blockcol[blockidx] = 172;
					else if (rand % 10 < 7) blockcol[blockidx] = 8;
					else blockcol[blockidx] = 9;

					if(rand%300==0) blockcol[blockidx] = 149;
					if (rand % 1000000 == 0)
					{
						blockcol[blockidx] = 0;
					}
				}
				else if (disttmp < 499)
				{
					blocks[blockidx] = true;
					blockcol[blockidx] = 180;

					if (k > 957) blockcol[blockidx] = 204;
					if (k > 960) blockcol[blockidx] = 215;

					if (k < 50) blockcol[blockidx] = 204;
					if (k < 48) blockcol[blockidx] = 215;
					

					if (tmp2 > 950) blockcol[blockidx] = 24;

					if (tmp2 < 75)blockcol[blockidx] = 18;

					if (j > 950) blockcol[blockidx] = 29;

					if (j < 75)blockcol[blockidx] = 101;
				}

				//if (k < 500) blocks[blockidx] = false;

}


__global__ void addKernel(uint8_t* buffer, double* vecl, bool* blocks, double* norm0, double* norm1, double* norm2, double* point0, double* point1, double* point2, int* mir, double pos0, double pos1, double pos2, double vec0, double vec1, double vec2, double addy0, double addy1, double addy2, double addz0, double addz1, double addz2, int nbblocks, uint8_t* blockcol, uint8_t* stars, int nbframe, int skyr, int skyg, int skyb, double* polp0, double* polp1, double* polp2, double* v0p0, double* v0p1, double* v0p2, double* v1p0, double* v1p1, double* v1p2, double dtmpmax)
{
	int i,l;

	double sp1 = 40000;
	double sp2 = 0.001;

	double vecn0, vecn1, vecn2;
	double cpos0, cpos1, cpos2;
	double tpos0, tpos1, tpos2;
	int cnx, cny, cnz;

	double tmin;
	double ttmp;
	double alpha;

	double px, py, pz;

	double qa, qb, qc;
	double discr;

	double t1, t2;
	double t1f, t2f;
	double tcont;

	double skyfac;
	double min;
	double tmpd;

	double kb, kc, kd;

	int col, colr, colg;
	int uv2;

	int tmp = blockIdx.x * blockDim.x + threadIdx.x;
	int tmpx = tmp % 1280;
	int tmpy = (tmp - tmpx) / 1280;

	int coll;
	int cnx2;
	int cface;

	double prj0, prj1;

	double tmpl, v0h0, v0h1, v0h2;
	double v1h0, v1h1, v1h2;
	double v2l;
	double v20, v21, v22;
	double spv0v1;
	double v2h0, v2h1, v2h2, p0, p1;

	int rnbwv;

	double dtmp;
	double sptmp;

	double u, v;

	double colp0, colp1, colp2;
	double tmpnorm0, tmpnorm1, tmpnorm2;

	int blockidx;
	uint8_t currblock;

	uint8_t uv;

	vecn0 = vec0 + tmpx * addy0 + tmpy * addz0;
	vecn1 = vec1 + tmpx * addy1 + tmpy * addz1;
	vecn2 = vec2 + tmpx * addy2 + tmpy * addz2;

	vecn0 /= vecl[tmp];
	vecn1 /= vecl[tmp];
	vecn2 /= vecl[tmp];


	qa = vecn0 * vecn0 + vecn1 * vecn1 + vecn2 * vecn2;
	qb = 2 * (vecn0*pos0+ vecn1 * pos1 + vecn2 * pos2 ) - 1000 * (vecn0+vecn1+vecn2);
	qc = pos0 * pos0 + pos1 * pos1 + pos2 * pos2 - 1000 * (pos0+pos1+pos2-500);

	discr = qb * qb - 4 * qa * qc;

	if (discr <= 0)
	{
		u = 2000*((0.5 + atan2(vecn1, vecn0) / (2.0 * M_PI)));
		v =2000*((0.5 + asin(vecn2) / M_PI));
		uv = stars[(int)u + 2000 * (int)v];

		qa = vecn0 * vecn0 + vecn1 * vecn1 + vecn2 * vecn2;
		qb = 2 * (vecn0 * pos0 + vecn1 * pos1 + vecn2 * pos2) - 1000 * (vecn0 + vecn1 + vecn2);
		qc = pos0 * pos0 + pos1 * pos1 + pos2 * pos2 - 1000 * (pos0 + pos1 + pos2) - 250000;

		discr = qb * qb - 4 * qa * qc;

		if (discr <= 0) skyfac = 0;
		else
		{
			t1 = ((-1) * qb - sqrt(discr)) / (2.0 * qa);
			t2 = ((-1) * qb + sqrt(discr)) / (2.0 * qa);

			if (t1 < 0 && t2 < 0) skyfac = 0;
			else
			{
				if (t2 > t1)
				{
					t1f = t1;
					t2f = t2;
				}
				else
				{
					t1f = t2;
					t2f = t1;
				}
				if (t1f < 0) t1f = 0;

				kb = vecn0 * vecn0 + vecn1 * vecn1 + vecn2 * vecn2;
				kc = 2 * vecn0 * pos0 - 1000 * vecn0;
				kc += 2 * vecn1 * pos1 - 1000 * vecn1;
				kc += 2 * vecn2 * pos2 - 1000 * vecn2;
				kd = pos0 * pos0 - 1000 * (pos0-250);
				kd += pos1 * pos1 - 1000 * (pos1 - 250);
				kd += pos2 * pos2 - 1000 * (pos2 - 250);

				skyfac = skyfunc(-2000 * sp2, kb, kc, kd, 1000000*sp2, sp2, t2f) - skyfunc(-2000*sp2, kb, kc, kd, 1000000*sp2,sp2, t1f);
				skyfac /= sp1;
				if (skyfac > 1)skyfac = 1;

			}
		}

		buffer[4 * tmp] = skyfac*skyr + (1-skyfac)* uv;
		buffer[4 * tmp + 1] = skyfac * skyg + (1 - skyfac) * uv;
		buffer[4 * tmp + 2] = skyfac * skyb + (1 - skyfac) * uv;
		buffer[4 * tmp + 3] = 255;
	}
	else
	{
		t1 = ((-1) * qb - sqrt(discr)) / (2.0 * qa);
		t2 = ((-1) * qb + sqrt(discr)) / (2.0 * qa);

		if (t1 < 0 && t2 < 0)
		{
			u = 2000 * ((0.5 + atan2(vecn1, vecn0) / (2.0 * M_PI)));
			v = 2000 * ((0.5 + asin(vecn2) / M_PI));
			uv = stars[(int)u + 2000 * (int)v];

			qa = vecn0 * vecn0 + vecn1 * vecn1 + vecn2 * vecn2;
			qb = 2 * (vecn0 * pos0 + vecn1 * pos1 + vecn2 * pos2) - 1000 * (vecn0 + vecn1 + vecn2);
			qc = pos0 * pos0 + pos1 * pos1 + pos2 * pos2 - 1000 * (pos0 + pos1 + pos2) - 250000;

			discr = qb * qb - 4 * qa * qc;

			if (discr <= 0) skyfac = 0;
			else
			{
				t1 = ((-1) * qb - sqrt(discr)) / (2.0 * qa);
				t2 = ((-1) * qb + sqrt(discr)) / (2.0 * qa);

				if (t1 < 0 && t2 < 0) skyfac = 0;
				else
				{
					if (t2 > t1)
					{
						t1f = t1;
						t2f = t2;
					}
					else
					{
						t1f = t2;
						t2f = t1;
					}
					if (t1f < 0) t1f = 0;

					kb = vecn0 * vecn0 + vecn1 * vecn1 + vecn2 * vecn2;
					kc = 2 * vecn0 * pos0 - 1000 * vecn0;
					kc += 2 * vecn1 * pos1 - 1000 * vecn1;
					kc += 2 * vecn2 * pos2 - 1000 * vecn2;
					kd = pos0 * pos0 - 1000 * (pos0 - 250);
					kd += pos1 * pos1 - 1000 * (pos1 - 250);
					kd += pos2 * pos2 - 1000 * (pos2 - 250);

					skyfac = skyfunc(-2000 * sp2, kb, kc, kd, 1000000*sp2, sp2, t2f) - skyfunc(-2000 * sp2, kb, kc, kd, 1000000*sp2, sp2, t1f);
					skyfac /= sp1;
					if (skyfac > 1)skyfac = 1;

				}
			}

			buffer[4 * tmp] = skyfac * skyr + (1 - skyfac) * uv;
			buffer[4 * tmp + 1] = skyfac * skyg + (1 - skyfac) * uv;
			buffer[4 * tmp + 2] = skyfac * skyb + (1 - skyfac) * uv;
			buffer[4 * tmp + 3] = 255;
		}
		else
		{
			if (t1 * t2 > 0)
			{
				if (t1 < t2) tcont = t1;
				else tcont = t2;

				cpos0 = pos0 + tcont * vecn0;
				cpos1 = pos1 + tcont * vecn1;
				cpos2 = pos2 + tcont * vecn2;
			}
			else
			{
				cpos0 = pos0;
				cpos1 = pos1;
				cpos2 = pos2;
			}

			px = fmod(cpos0, 1.0);
			py = fmod(cpos1, 1.0);
			pz = fmod(cpos2, 1.0);

			if (px < 0) px++;
			if (py < 0) py++;
			if (pz < 0) pz++;

			cnx = cpos0 - px;
			cny = cpos1 - py;
			cnz = cpos2 - pz;


			if ((cnx + cny + cnz) % 2 != 0)
			{
				if (px < py)
				{
					if (px < pz)
					{
						if (px < 1 - py)
						{
							if (px < 1 - pz)
							{
								px++;
								cnx--;
							}
							else
							{
								pz--;
								cnz++;
							}
						}
						else
						{
							if (1 - py < 1 - pz)
							{
								py--;
								cny++;
							}
							else
							{
								pz--;
								cnz++;
							}
						}
					}
					else
					{
						if (pz < 1 - py)
						{
							pz++;
							cnz--;
						}
						else
						{
							py--;
							cny++;
						}
					}
				}
				else
				{
					if (py < pz)
					{
						if (py < 1 - px)
						{
							if (py < 1 - pz)
							{
								py++;
								cny--;
							}
							else
							{
								pz--;
								cnz++;
							}
						}
						else
						{
							if (1 - px < 1 - pz)
							{
								px--;
								cnx++;
							}
							else
							{
								pz--;
								cnz++;
							}
						}
					}
					else
					{
						if (pz < 1 - px)
						{
							pz++;
							cnz--;
						}
						else
						{
							px--;
							cnx++;
						}
					}
				}
			}

			if (cnx < 0 || cnx >= nbblocks || cny < 0 || cny >= nbblocks || cnz < 0 || cnz >= nbblocks)
			{
				col = 1;
			}
			else
			{
				if (cnx % 2 != 0) cnx2 = (cnx - 1) / 2;
				else cnx2 = cnx / 2;

				blockidx = cnx2 + nbblocks * cny + nbblocks * nbblocks * cnz;

				col = blockcol[blockidx];
			}

			if (blocks[blockidx] && col == 0)
			{
				cpos0 = px;
				cpos1 = py;
				cpos2 = pz;

				dtmp = 0;
				cface = -1;
				tmpd = 0;
				while (dtmp < dtmpmax)
				{
					tmin = 4.0;


					for (i = 0; i < 12; i++)
					{
						if (i != cface) {
							ttmp = (point0[i] - cpos0) * norm0[i] + (point1[i] - cpos1) * norm1[i] + (point2[i] - cpos2) * norm2[i];
							ttmp /= norm0[i] * vecn0 + norm1[i] * vecn1 + norm2[i] * vecn2;

							if (ttmp > 0 && ttmp < tmin)
							{
								tmin = ttmp;
								coll = i;
							}
						}
					}

					dtmp += tmin;

					if (dtmp < dtmpmax) {
						cpos0 += tmin * vecn0;
						cpos1 += tmin * vecn1;
						cpos2 += tmin * vecn2;

						tmpnorm0 = norm0[coll] / sqrt(2.0);
						tmpnorm1 = norm1[coll] / sqrt(2.0);
						tmpnorm2 = norm2[coll] / sqrt(2.0);

						sptmp = vecn0 * tmpnorm0 + vecn1 * tmpnorm1 + vecn2 * tmpnorm2;

						vecn0 = vecn0 - 2 * sptmp * tmpnorm0;
						vecn1 = vecn1 - 2 * sptmp * tmpnorm1;
						vecn2 = vecn2 - 2 * sptmp * tmpnorm2;

						cface = coll;

						///////

						colp0 = cpos0;
						colp1 = cpos1;
						colp2 = cpos2;

						colp0 -= polp0[cface];
						colp1 -= polp1[cface];
						colp2 -= polp2[cface];

						tmpl = sqrt(v0p0[cface] * v0p0[cface] + v0p1[cface] * v0p1[cface] + v0p2[cface] * v0p2[cface]);

						v0h0 = v0p0[cface] / tmpl;
						v0h1 = v0p1[cface] / tmpl;
						v0h2 = v0p2[cface] / tmpl;

						v1h0 = v1p0[cface] / tmpl;
						v1h1 = v1p1[cface] / tmpl;
						v1h2 = v1p2[cface] / tmpl;

						spv0v1 = v0h0 * v1h0 + v0h1 * v1h1 + v0h2 * v1h2;

						v20 = v1h0 - spv0v1 * v0h0;
						v21 = v1h1 - spv0v1 * v0h1;
						v22 = v1h2 - spv0v1 * v0h2;

						v2l = sqrt(v20 * v20 + v21 * v21 + v22 * v22);

						v2h0 = v20 / v2l;
						v2h1 = v21 / v2l;
						v2h2 = v22 / v2l;

						p0 = colp0 * v0h0 + colp1 * v0h1 + colp2 * v0h2;
						p1 = colp0 * v2h0 + colp1 * v2h1 + colp2 * v2h2;

						prj0 = (p0 - (p1 * spv0v1 / v2l)) / tmpl;
						prj1 = p1 / (v2l * tmpl);

						if (prj0 < prj1) min = prj0;
						else min = prj1;
						if (1 - prj0 < min) min = 1 - prj0;
						if (1 - prj1 < min) min = 1 - prj1;

						tmpd += (1 - dtmp / dtmpmax) * (gauss(min) / gauss(0));
					}
				}
				/////
				if (tmpd > 1) tmpd = 1;

				if (tmpd < 0.5)
				{
					colg = 0;
					colr = (int)(510.0 * tmpd);
					col = (int)(510.0 * tmpd);
				}
				else
				{
					colr = 255;
					col = 255;
					colg = (int)(510.0 * tmpd - 255.0);
				}


				buffer[4 * tmp] = colr;
				buffer[4 * tmp + 1] = colg;
				buffer[4 * tmp + 2] = col;
				buffer[4 * tmp + 3] = 255;



				return;
			}

			tpos0 = cpos0;
			tpos1 = cpos1;
			tpos2 = cpos2;

			cpos0 = px;
			cpos1 = py;
			cpos2 = pz;

			tmin = 4;


			for (i = 0; i < 12; i++)
			{
				ttmp = (point0[i] - cpos0) * norm0[i] + (point1[i] - cpos1) * norm1[i] + (point2[i] - cpos2) * norm2[i];
				ttmp /= norm0[i] * vecn0 + norm1[i] * vecn1 + norm2[i] * vecn2;

				if (ttmp > 0 && ttmp < tmin)
				{
					tmin = ttmp;
					coll = i;
				}
			}

			

			cnx -= norm0[coll];
			cny -= norm1[coll];
			cnz -= norm2[coll];

			cpos0 += tmin * vecn0;
			cpos1 += tmin * vecn1;
			cpos2 += tmin * vecn2;

			tpos0 += tmin * vecn0;
			tpos1 += tmin * vecn1;
		    tpos2 += tmin * vecn2;

			cpos0 += norm0[coll];
			cpos1 += norm1[coll];
			cpos2 += norm2[coll];

			cface = mir[coll];

			if (cnx < 0 || cnx >= nbblocks || cny < 0 || cny >= nbblocks || cnz < 0 || cnz >= nbblocks)
			{
				currblock = 0;
			}
			else
			{
				if (cnx % 2 != 0) cnx2 = (cnx - 1) / 2;
				else cnx2 = cnx / 2;

				blockidx = cnx2 + nbblocks * cny + nbblocks * nbblocks * cnz;

				currblock = blocks[blockidx];
			}

			while (!currblock && sqrt((tpos0-500.0)*(tpos0-500.0)+(tpos1-500.0)*(tpos1-500.0)+(tpos2-500.0)*(tpos2-500.0)) < 500.0)
			{
				tmin = 4.0;


				for (i = 0; i < 12; i++)
				{
					if (i != cface) {
						ttmp = (point0[i] - cpos0) * norm0[i] + (point1[i] - cpos1) * norm1[i] + (point2[i] - cpos2) * norm2[i];
						ttmp /= norm0[i] * vecn0 + norm1[i] * vecn1 + norm2[i] * vecn2;

						if (ttmp > 0 && ttmp < tmin)
						{
							tmin = ttmp;
							coll = i;
						}
					}
				}

				cnx -= norm0[coll];
				cny -= norm1[coll];
				cnz -= norm2[coll];

				cpos0 += tmin * vecn0;
				cpos1 += tmin * vecn1;
				cpos2 += tmin * vecn2;

				tpos0 += tmin * vecn0;
				tpos1 += tmin * vecn1;
				tpos2 += tmin * vecn2;

				cpos0 += norm0[coll];
				cpos1 += norm1[coll];
				cpos2 += norm2[coll];

				cface = mir[coll];

				if (cnx < 0 || cnx >= nbblocks || cny < 0 || cny >= nbblocks || cnz < 0 || cnz >= nbblocks)
				{
					currblock = false;
				}
				else
				{
					if (cnx % 2 != 0) cnx2 = (cnx - 1) / 2;
					else cnx2 = cnx / 2;
					cnx2 = cnx/2;

					blockidx = cnx2 + nbblocks * cny + nbblocks * nbblocks * cnz;

					currblock = blocks[blockidx];
				}
			}

			
			if (currblock)
			{
				col = blockcol[blockidx];
				if (col == 255)
				{
					rnbwv = rnbw(nbframe);

					colr = rnbwv % 256;
					rnbwv -= colr;
					rnbwv /= 256;
					colg = rnbwv % 256;
					rnbwv -= colg;
					rnbwv /= 256;
					col = rnbwv % 256;

					alpha = 1 - 0.042 * cface;
					buffer[4 * tmp] = colr * alpha;
					buffer[4 * tmp + 1] = colg * alpha;
					buffer[4 * tmp + 2] = col * alpha;
					buffer[4 * tmp + 3] = 255;
				}
				else if (col == 0)
				{
					dtmp = 0;

					colp0 = cpos0;
					colp1 = cpos1;
					colp2 = cpos2;

					colp0 -= polp0[cface];
					colp1 -= polp1[cface];
					colp2 -= polp2[cface];

					tmpl = sqrt(v0p0[cface] * v0p0[cface] + v0p1[cface] * v0p1[cface] + v0p2[cface] * v0p2[cface]);

					v0h0 = v0p0[cface] / tmpl;
					v0h1 = v0p1[cface] / tmpl;
					v0h2 = v0p2[cface] / tmpl;

					v1h0 = v1p0[cface] / tmpl;
					v1h1 = v1p1[cface] / tmpl;
					v1h2 = v1p2[cface] / tmpl;

					spv0v1 = v0h0 * v1h0 + v0h1 * v1h1 + v0h2 * v1h2;

					v20 = v1h0 - spv0v1 * v0h0;
					v21 = v1h1 - spv0v1 * v0h1;
					v22 = v1h2 - spv0v1 * v0h2;

					v2l = sqrt(v20 * v20 + v21 * v21 + v22 * v22);

					v2h0 = v20 / v2l;
					v2h1 = v21 / v2l;
					v2h2 = v22 / v2l;

					p0 = colp0 * v0h0 + colp1 * v0h1 + colp2 * v0h2;
					p1 = colp0 * v2h0 + colp1 * v2h1 + colp2 * v2h2;

					prj0 = (p0 - (p1 * spv0v1 / v2l)) / tmpl;
					prj1 = p1 / (v2l * tmpl);

					if (prj0 < prj1) min = prj0;
					else min = prj1;
					if (1 - prj0 < min) min = 1 - prj0;
					if (1 - prj1 < min) min = 1 - prj1;

					tmpd = gauss(min) / gauss(0);

					/////
					while (dtmp<dtmpmax)
					{
						tmin = 4.0;


						for (i = 0; i < 12; i++)
						{
							if (i != cface) {
								ttmp = (point0[i] - cpos0) * norm0[i] + (point1[i] - cpos1) * norm1[i] + (point2[i] - cpos2) * norm2[i];
								ttmp /= norm0[i] * vecn0 + norm1[i] * vecn1 + norm2[i] * vecn2;

								if (ttmp > 0 && ttmp < tmin)
								{
									tmin = ttmp;
									coll = i;
								}
							}
						}

						dtmp += tmin;

						if (dtmp < dtmpmax) {
							cpos0 += tmin * vecn0;
							cpos1 += tmin * vecn1;
							cpos2 += tmin * vecn2;

							tmpnorm0 = norm0[coll] / sqrt(2.0);
							tmpnorm1 = norm1[coll] / sqrt(2.0);
							tmpnorm2 = norm2[coll] / sqrt(2.0);

							sptmp = vecn0 * tmpnorm0 + vecn1 * tmpnorm1 + vecn2 * tmpnorm2;

							vecn0 = vecn0 - 2 * sptmp * tmpnorm0;
							vecn1 = vecn1 - 2 * sptmp * tmpnorm1;
							vecn2 = vecn2 - 2 * sptmp * tmpnorm2;

							cface = coll;

							///////

							colp0 = cpos0;
							colp1 = cpos1;
							colp2 = cpos2;

							colp0 -= polp0[cface];
							colp1 -= polp1[cface];
							colp2 -= polp2[cface];

							tmpl = sqrt(v0p0[cface] * v0p0[cface] + v0p1[cface] * v0p1[cface] + v0p2[cface] * v0p2[cface]);

							v0h0 = v0p0[cface] / tmpl;
							v0h1 = v0p1[cface] / tmpl;
							v0h2 = v0p2[cface] / tmpl;

							v1h0 = v1p0[cface] / tmpl;
							v1h1 = v1p1[cface] / tmpl;
							v1h2 = v1p2[cface] / tmpl;

							spv0v1 = v0h0 * v1h0 + v0h1 * v1h1 + v0h2 * v1h2;

							v20 = v1h0 - spv0v1 * v0h0;
							v21 = v1h1 - spv0v1 * v0h1;
							v22 = v1h2 - spv0v1 * v0h2;

							v2l = sqrt(v20 * v20 + v21 * v21 + v22 * v22);

							v2h0 = v20 / v2l;
							v2h1 = v21 / v2l;
							v2h2 = v22 / v2l;

							p0 = colp0 * v0h0 + colp1 * v0h1 + colp2 * v0h2;
							p1 = colp0 * v2h0 + colp1 * v2h1 + colp2 * v2h2;

							prj0 = (p0 - (p1 * spv0v1 / v2l)) / tmpl;
							prj1 = p1 / (v2l * tmpl);

							if (prj0 < prj1) min = prj0;
							else min = prj1;
							if (1 - prj0 < min) min = 1 - prj0;
							if (1 - prj1 < min) min = 1 - prj1;

							tmpd += (1 - dtmp / dtmpmax) * (gauss(min) / gauss(0));
						}
					}
					/////
					if (tmpd > 1) tmpd = 1;

					if (tmpd < 0.5)
					{
						colg = 0;
						colr = (int)(510.0 * tmpd);
						col = (int)(510.0 * tmpd);
					}
					else
					{
						colr = 255;
						col = 255;
						colg = (int)(510.0 * tmpd - 255.0);
					}


					buffer[4 * tmp] = colr;
					buffer[4 * tmp + 1] = colg;
					buffer[4 * tmp + 2] = col;
					buffer[4 * tmp + 3] = 255;
				}
				else
				{
					colr = col % 6;
					col -= colr;
					col /= 6;
					colg = col % 6;
					col -= colg;
					col /= 6;
					col %= 6;

					colr *= (255 / 5);
					colg *= (255 / 5);
					col *= (255 / 5);

					//
					colp0 = cpos0;
					colp1 = cpos1;
					colp2 = cpos2;

					colp0 -= polp0[cface];
					colp1 -= polp1[cface];
					colp2 -= polp2[cface];

					tmpl = sqrt(v0p0[cface] * v0p0[cface] + v0p1[cface] * v0p1[cface] + v0p2[cface] * v0p2[cface]);

					v0h0 = v0p0[cface] / tmpl;
					v0h1 = v0p1[cface] / tmpl;
					v0h2 = v0p2[cface] / tmpl;

					v1h0 = v1p0[cface] / tmpl;
					v1h1 = v1p1[cface] / tmpl;
					v1h2 = v1p2[cface] / tmpl;

					spv0v1 = v0h0 * v1h0 + v0h1 * v1h1 + v0h2 * v1h2;

					v20 = v1h0 - spv0v1 * v0h0;
					v21 = v1h1 - spv0v1 * v0h1;
					v22 = v1h2 - spv0v1 * v0h2;
						
					v2l = sqrt(v20*v20+v21*v21+v22*v22);

					v2h0 = v20 / v2l;
					v2h1 = v21 / v2l;
					v2h2 = v22 / v2l;

					p0 = colp0 * v0h0 + colp1 * v0h1 + colp2 * v0h2;
					p1 = colp0 * v2h0 + colp1 * v2h1 + colp2 * v2h2;

					prj0 = (p0 - (p1 * spv0v1 / v2l)) / tmpl;
					prj1 = p1 / (v2l*tmpl);
					

					u = 10 * prj0;
					v = 10 * prj1;
					uv2 = (int)u + 10 * (int)v+cface;


					for (l = 0; l < 10; l++) uv2 = (60493 * uv2 + 11) % 479001599;

					uv2 %= 6;
					uv2 -= 3;
					//
					
					alpha = 1 - 0.05 * cface;
					colr *= alpha;
					colg *= alpha;
					col *= alpha;

					colr += uv2;
					colg += uv2;
					col += uv2;
					if (colr < 0) colr = 0;
					else if (colr > 255) colr = 255;
					if (colg < 0) colg = 0;
					else if (colg > 255) colg = 255;
					if (col < 0) col = 0;
					else if (col > 255) col = 255;

					buffer[4 * tmp] = colr;
					buffer[4 * tmp + 1] = colg;
					buffer[4 * tmp + 2] = col;
					buffer[4 * tmp + 3] = 255;
				}
			}
			else
			{
				buffer[4 * tmp] = skyr;
				buffer[4 * tmp + 1] = skyg;
				buffer[4 * tmp + 2] = skyb;
				buffer[4 * tmp + 3] = 255;
			}
		}
	}
}


void cudaInit(bool* blockstmp)
{
	double dist = 2;
	double sqsz = 0.01 / 4;
	int tmpx, tmpy;

	double* vecltmp = new double[1280 * 720];

	double vec0, vec1, vec2;
	double addy0, addy1, addy2;
	double addz0, addz1, addz2;
	double vecn0, vecn1, vecn2;
	double x00 = 1, x01 = 0, x02 = 0;
	double x10 = 0, x11 = 1, x12 = 0;
	double x20 = 0, x21 = 0, x22 = 1;
	double multy = (1 - 1280) * sqsz / 2;
	double multz = (720 - 1) * sqsz / 2;

	double* norm0tmp = new double[12];
	double* norm1tmp = new double[12];
	double* norm2tmp = new double[12];
	double* point0tmp = new double[12];
	double* point1tmp = new double[12];
	double* point2tmp = new double[12];

	double polp0tmp[12]{};
	double polp1tmp[12]{};
	double polp2tmp[12]{};

	double v0p0tmp[12]{};
	double v0p1tmp[12]{};
	double v0p2tmp[12]{};

	double v1p0tmp[12]{};
	double v1p1tmp[12]{};
	double v1p2tmp[12]{};

	int* mirtmp = new int[12];

	uint8_t* blockcoltmp = new uint8_t[nbblocks * nbblocks * nbblocks];
	uint8_t* starstmp = new uint8_t[2000 * 2000];


	hipSetDevice(0);
	hipMalloc((void**)&buffer, 4 * 1280 * 720 * sizeof(uint8_t));
	hipMalloc((void**)&vecl, 1280 * 720 * sizeof(double));
	hipMalloc((void**)&blocks, nbblocks * nbblocks * nbblocks * sizeof(bool));

	hipMalloc((void**)&norm0, 12 * sizeof(double));
	hipMalloc((void**)&norm1, 12 * sizeof(double));
	hipMalloc((void**)&norm2, 12 * sizeof(double));

	hipMalloc((void**)&point0, 12 * sizeof(double));
	hipMalloc((void**)&point1, 12 * sizeof(double));
	hipMalloc((void**)&point2, 12 * sizeof(double));

	hipMalloc((void**)&polp0, 12 * sizeof(double));
	hipMalloc((void**)&polp1, 12 * sizeof(double));
	hipMalloc((void**)&polp2, 12 * sizeof(double));

	hipMalloc((void**)&v0p0, 12 * sizeof(double));
	hipMalloc((void**)&v0p1, 12 * sizeof(double));
	hipMalloc((void**)&v0p2, 12 * sizeof(double));

	hipMalloc((void**)&v1p0, 12 * sizeof(double));
	hipMalloc((void**)&v1p1, 12 * sizeof(double));
	hipMalloc((void**)&v1p2, 12 * sizeof(double));

	hipMalloc((void**)&mir, 12 * sizeof(int));

	hipMalloc((void**)&blockcol, nbblocks*nbblocks*nbblocks * sizeof(uint8_t));

	hipMalloc((void**)&stars, 2000*2000* sizeof(uint8_t));

	vec0 = dist * x00 + multy * x10 + multz * x20;
	vec1 = dist * x01 + multy * x11 + multz * x21;
	vec2 = dist * x02 + multy * x12 + multz * x22;

	addy0 = sqsz * x10;
	addy1 = sqsz * x11;
	addy2 = sqsz * x12;

	addz0 = -sqsz * x20;
	addz1 = -sqsz * x21;
	addz2 = -sqsz * x22;

	for (int i = 0; i < 1280 * 720; i++)
	{
		tmpx = i % 1280;
		tmpy = (i - tmpx) / 1280;

		vecn0 = vec0 + tmpx * addy0 + tmpy * addz0;
		vecn1 = vec1 + tmpx * addy1 + tmpy * addz1;
		vecn2 = vec2 + tmpx * addy2 + tmpy * addz2;

		vecltmp[i] = sqrt(vecn0 * vecn0 + vecn1 * vecn1 + vecn2 * vecn2);
	}



	point0tmp[0] = 0.5;
	point1tmp[0] = 0.5;
	point2tmp[0] = 1.5;

	point0tmp[1] = 0.5;
	point1tmp[1] = 0.5;
	point2tmp[1] = 1.5;

	point0tmp[2] = 0.5;
	point1tmp[2] = 0.5;
	point2tmp[2] = 1.5;

	point0tmp[3] = 0.5;
	point1tmp[3] = 0.5;
	point2tmp[3] = 1.5;

	point0tmp[4] = 1.5;
	point1tmp[4] = 0.5;
	point2tmp[4] = 0.5;

	point0tmp[5] = 0.5;
	point1tmp[5] = 1.5;
	point2tmp[5] = 0.5;

	point0tmp[6] = -0.5;
	point1tmp[6] = 0.5;
	point2tmp[6] = 0.5;

	point0tmp[7] = 0.5;
	point1tmp[7] = -0.5;
	point2tmp[7] = 0.5;

	point0tmp[8] = 0.5;
	point1tmp[8] = 0.5;
	point2tmp[8] = -0.5;

	point0tmp[9] = 0.5;
	point1tmp[9] = 0.5;
	point2tmp[9] = -0.5;

	point0tmp[10] = 0.5;
	point1tmp[10] = 0.5;
	point2tmp[10] = -0.5;

	point0tmp[11] = 0.5;
	point1tmp[11] = 0.5;
	point2tmp[11] = -0.5;

	norm0tmp[0] = -1;
	norm1tmp[0] = 0;
	norm2tmp[0] = -1;

	norm0tmp[1] = 0;
	norm1tmp[1] = -1;
	norm2tmp[1] = -1;

	norm0tmp[2] = 1;
	norm1tmp[2] = 0;
	norm2tmp[2] = -1;

	norm0tmp[3] = 0;
	norm1tmp[3] = 1;
	norm2tmp[3] = -1;

	norm0tmp[4] = -1;
	norm1tmp[4] = -1;
	norm2tmp[4] = 0;

	norm0tmp[5] = 1;
	norm1tmp[5] = -1;
	norm2tmp[5] = 0;

	norm0tmp[6] = 1;
	norm1tmp[6] = 1;
	norm2tmp[6] = 0;

	norm0tmp[7] = -1;
	norm1tmp[7] = 1;
	norm2tmp[7] = 0;

	norm0tmp[8] = -1;
	norm1tmp[8] = 0;
	norm2tmp[8] = 1;

	norm0tmp[9] = 0;
	norm1tmp[9] = -1;
	norm2tmp[9] = 1;

	norm0tmp[10] = 1;
	norm1tmp[10] = 0;
	norm2tmp[10] = 1;

	norm0tmp[11] = 0;
	norm1tmp[11] = 1;
	norm2tmp[11] = 1;

	mirtmp[0] = 10;
	mirtmp[1] = 11;
	mirtmp[2] = 8;
	mirtmp[3] = 9;
	mirtmp[4] = 6;
	mirtmp[5] = 7;
	mirtmp[6] = 4;
	mirtmp[7] = 5;
	mirtmp[8] = 2;
	mirtmp[9] = 3;
	mirtmp[10] = 0;
	mirtmp[11] = 1;
	
	polp0tmp[0] = 0.5;
	polp1tmp[0] = 0.5;
	polp2tmp[0] = 1.5;

	v0p0tmp[0] = 0.5;
	v0p1tmp[0] = 0.5;
	v0p2tmp[0] = -0.5;
	
	v1p0tmp[0] = 0.5;
	v1p1tmp[0] = -0.5;
	v1p2tmp[0] = -0.5;

	
	polp0tmp[1] = 0.5;
	polp1tmp[1] = 0.5;
	polp2tmp[1] = 1.5;
	
	v0p0tmp[1] = 0.5;
	v0p1tmp[1] = 0.5;
	v0p2tmp[1] = -0.5;

	v1p0tmp[1] = -0.5;
	v1p1tmp[1] = 0.5;
	v1p2tmp[1] = -0.5;
	
	polp0tmp[2] = 0.5;
	polp1tmp[2] = 0.5;
	polp2tmp[2] = 1.5;

	v0p0tmp[2] = -0.5;
	v0p1tmp[2] = -0.5;
	v0p2tmp[2] = -0.5;

	v1p0tmp[2] = -0.5;
	v1p1tmp[2] = 0.5;
	v1p2tmp[2] = -0.5;
	
	polp0tmp[3] = 0.5;
	polp1tmp[3] = 0.5;
	polp2tmp[3] = 1.5;

	v0p0tmp[3] = -0.5;
	v0p1tmp[3] = -0.5;
	v0p2tmp[3] = -0.5;

	v1p0tmp[3] = 0.5;
	v1p1tmp[3] = -0.5;
	v1p2tmp[3] = -0.5;
	
	polp0tmp[4] = 1.5;
	polp1tmp[4] = 0.5;
	polp2tmp[4] = 0.5;

	v0p0tmp[4] = -0.5;
	v0p1tmp[4] = 0.5;
	v0p2tmp[4] = 0.5;

	v1p0tmp[4] = -0.5;
	v1p1tmp[4] = 0.5;
	v1p2tmp[4] = -0.5;
	

	polp0tmp[5] = 0.5;
	polp1tmp[5] = 1.5;
	polp2tmp[5] = 0.5;

	v0p0tmp[5] = -0.5;
	v0p1tmp[5] = -0.5;
	v0p2tmp[5] = 0.5;

	v1p0tmp[5] = -0.5;
	v1p1tmp[5] = -0.5;
	v1p2tmp[5] = -0.5;
	

	polp0tmp[6] = -0.5;
	polp1tmp[6] = 0.5;
	polp2tmp[6] = 0.5;

	v0p0tmp[6] = 0.5;
	v0p1tmp[6] = -0.5;
	v0p2tmp[6] = 0.5;

	v1p0tmp[6] = 0.5;
	v1p1tmp[6] = -0.5;
	v1p2tmp[6] = -0.5;
	

	polp0tmp[7] = 0.5;
	polp1tmp[7] = -0.5;
	polp2tmp[7] = 0.5;

	v0p0tmp[7] = 0.5;
	v0p1tmp[7] = 0.5;
	v0p2tmp[7] = 0.5;

	v1p0tmp[7] = 0.5;
	v1p1tmp[7] = 0.5;
	v1p2tmp[7] = -0.5;
	

	polp0tmp[8] = 0.5;
	polp1tmp[8] = 0.5;
	polp2tmp[8] = -0.5;

	v0p0tmp[8] = 0.5;
	v0p1tmp[8] = -0.5;
	v0p2tmp[8] = 0.5;

	v1p0tmp[8] = 0.5;
	v1p1tmp[8] = 0.5;
	v1p2tmp[8] = 0.5;
	

	polp0tmp[9] = 0.5;
	polp1tmp[9] = 0.5;
	polp2tmp[9] = -0.5;

	v0p0tmp[9] = -0.5;
	v0p1tmp[9] = 0.5;
	v0p2tmp[9] = 0.5;

	v1p0tmp[9] = 0.5;
	v1p1tmp[9] =0.5;
	v1p2tmp[9] = 0.5;
	

	polp0tmp[10] = 0.5;
	polp1tmp[10] = 0.5;
	polp2tmp[10] = -0.5;

	v0p0tmp[10] = -0.5;
	v0p1tmp[10] = -0.5;
	v0p2tmp[10] = 0.5;

	v1p0tmp[10] = -0.5;
	v1p1tmp[10] = 0.5;
	v1p2tmp[10] = 0.5;
	

	polp0tmp[11] = 0.5;
	polp1tmp[11] = 0.5;
	polp2tmp[11] = -0.5;

	v0p0tmp[11] = -0.5;
	v0p1tmp[11] = -0.5;
	v0p2tmp[11] = 0.5;

	v1p0tmp[11] = 0.5;
	v1p1tmp[11] = -0.5;
	v1p2tmp[11] = 0.5;

	hipMemcpy(vecl, vecltmp, 1280 * 720 * sizeof(double), hipMemcpyHostToDevice);
	
	hipMemcpy(norm0, norm0tmp, 12 * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(norm1, norm1tmp, 12 * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(norm2, norm2tmp, 12 * sizeof(double), hipMemcpyHostToDevice);

	hipMemcpy(point0, point0tmp, 12 * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(point1, point1tmp, 12 * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(point2, point2tmp, 12 * sizeof(double), hipMemcpyHostToDevice);

	hipMemcpy(polp0, polp0tmp, 12 * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(polp1, polp1tmp, 12 * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(polp2, polp2tmp, 12 * sizeof(double), hipMemcpyHostToDevice);

	hipMemcpy(v0p0, v0p0tmp, 12 * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(v0p1, v0p1tmp, 12 * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(v0p2, v0p2tmp, 12 * sizeof(double), hipMemcpyHostToDevice);

	hipMemcpy(v1p0, v1p0tmp, 12 * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(v1p1, v1p1tmp, 12 * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(v1p2, v1p2tmp, 12 * sizeof(double), hipMemcpyHostToDevice);


	hipMemcpy(mir, mirtmp, 12 * sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy(stars, starstmp,2000*2000 * sizeof(uint8_t), hipMemcpyHostToDevice);

	setstars << <2000 * 2000 / 500, 500 >> > (stars);
	hipDeviceSynchronize();

	setplanet << <nbblocks*nbblocks*nbblocks / 500, 500 >> > (blocks,blockcol);
	hipDeviceSynchronize();

	hipMemcpy(blockstmp, blocks, nbblocks * nbblocks * nbblocks * sizeof(bool), hipMemcpyDeviceToHost);

}

void cudaExit()
{
	hipFree(buffer);
	hipFree(vecl);
	hipDeviceReset();
}

void cudathingy(uint8_t* pixels, double pos0, double pos1, double pos2, double vec0, double vec1, double vec2, double addy0, double addy1, double addy2, double addz0, double addz1, double addz2, int remidx, int addidx, int buildidx, uint8_t col, int nbframe, double dtmpmax)
{
	if (remidx != -1)
	{
		remblock << <1, 1 >> > (blocks,remidx);
		hipDeviceSynchronize();
	}
	if (addidx != -1)
	{
		addblock << <1, 1 >> > (blocks, addidx);
		hipDeviceSynchronize();
		
	}
	if (buildidx != -1)
	{
		changecol << <1, 1 >> > (blockcol, buildidx,col);
		hipDeviceSynchronize();
	}

	if (sqrt((pos0 - 500) * (pos0 - 500) + (pos1 - 500) * (pos1 - 500) + (pos2 - 500) * (pos2 - 500)) < 32)
	{
		skyr = 255;
		skyg = 174;
		skyb = 201;
	}

	addKernel <<<(int)(1280 * 720 / 600), 600 >>> (buffer, vecl, blocks, norm0, norm1, norm2, point0, point1, point2, mir, pos0, pos1, pos2, vec0, vec1, vec2, addy0, addy1, addy2, addz0, addz1, addz2, nbblocks, blockcol,stars,nbframe,skyr,skyg,skyb,polp0,polp1,polp2,v0p0,v0p1,v0p2,v1p0,v1p1,v1p2,dtmpmax);
	hipDeviceSynchronize();
	hipMemcpy(pixels, buffer, 4 * 1280 * 720 * sizeof(uint8_t), hipMemcpyDeviceToHost);
}