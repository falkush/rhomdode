#include "hip/hip_runtime.h"
﻿#define _USE_MATH_DEFINES

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>

static int nbblocks = 1000;

static uint8_t* buffer = 0;
static double* vecl = 0;
static bool* blocks = 0;
static uint8_t* blockcol = 0;

static double* norm0 = 0;
static double* norm1 = 0;
static double* norm2 = 0;

static double* point0 = 0;
static double* point1 = 0;
static double* point2 = 0;

static int* mir = 0;

static uint8_t* stars = 0;

static int skyr = 96;
static int skyg = 149;
static int skyb = 217;

__device__ double skyfunc(double a, double b, double c, double d, double e, double f, double x)
{
	return e*x+(a*(2.0*b*x+c)*sqrt(x*(b*x+c)+d))/(4.0*b)-(a*(c*c-4.0*b*d)*log(2.0*sqrt(b)*sqrt(x*(b*x+c)+d)+2.0*b*x+c))/(8.0*sqrt(b)*sqrt(b)*sqrt(b))+f*b*x*x*x/3.0+f*c*x*x/2+f*d*x;
}

__device__ int rnbw(int nbframe)
{
	int r=0, g=0, b=0;
	double tmp;
	double x;
	
		x = fmod(nbframe * 0.006, 1.0);
		tmp = fmod(x, 1.0 / 6.0);

		if (x < 1.0/6.0)
		{
			r = 255;
			g = 1530 * tmp;
		}
		else if (x < 1.0/3.0)
		{
			g = 255;
			r = 255 - 1530 * tmp;
		}
		else if (x < 0.5)
		{
			g = 255;
			b = 1530 * tmp;
		}
		else if (x < 2.0 / 3.0)
		{
			b = 255;
			g =255 - 1530 * tmp;
		}
		else if (x < 5.0 / 6.0)
		{
			b = 255;
			r = 1530 * tmp;
		}
		else
		{
			r = 255;
			b = 255 - 1530 * tmp;
		}

		return r+256*g+256*256*b;
}

__global__ void remblock(bool* blocks,int remidx)
{
	blocks[remidx] = false;
}

__global__ void addblock(bool* blocks, int addidx)
{
	blocks[addidx] = true;
}

__global__ void changecol (uint8_t* blockcol, int buildidx, uint8_t col)
{
	blockcol[buildidx] = col;
}

__global__ void setstars(uint8_t* stars)
{
	int i;
	int tmp = blockIdx.x * blockDim.x + threadIdx.x;

	int rand = tmp;

	for (i = 0; i < 10; i++) rand = (60493 *rand+11)% 115249;

	if ((rand)%200==0)
	{
		stars[tmp] = 255*rand/ 115249;
	}
	else
	{
		stars[tmp] = 0;
	}
}

__global__ void setplanet(bool* blocks, uint8_t* blockcol)
{
	int tmp, i, j, k,l;
	int rand;


	tmp = blockIdx.x * blockDim.x + threadIdx.x;
	rand = tmp;
	i = tmp % 500;
	tmp -= i;
	tmp /= 500;
	j = tmp%1000;
	tmp -= j;
	k = tmp / 1000;

	int blockidx;
	int nbblocks = 1000;
	double disttmp;
	int tmp2;
	int tmp3;

	

	for (l = 0; l < 100; l++) rand = (60493 * rand + 11) % 479001599;

				if ((j + k) % 2 == 0) tmp2 = 2 * i;
				else tmp2 = 2 * i + 1;
				
				disttmp = sqrt((tmp2 - 500 + 0.5) * (tmp2 - 500 + 0.5) + (j - 500 + 0.5) * (j - 500 + 0.5) + (k - 500 + 0.5) * (k - 500 + 0.5));
				blockidx = i + nbblocks * j + nbblocks * nbblocks * k;



				if (disttmp < 32)
				{
					blocks[blockidx] = true;
					blockcol[blockidx] = 255;
				}
				else if (disttmp < 64)
				{
					blocks[blockidx] = true;
					blockcol[blockidx] = 215;
				}
				else if (disttmp < 120)
				{
					blocks[blockidx] = true;
					if(rand%8==0) blockcol[blockidx] = 65;
					else if(rand%8==1 || rand%8==2) blockcol[blockidx] = 29;
					else blockcol[blockidx] = 35;
				}
				else if (disttmp < 180)
				{
					blocks[blockidx] = true;
					if (rand % 6 == 0) blockcol[blockidx] = 9;
					else if (rand % 6 == 1) blockcol[blockidx] = 11;
					else if (rand % 6 == 2) blockcol[blockidx] = 16;
					else blockcol[blockidx] = 17;
				}
				else if (disttmp < 250)
				{
					blocks[blockidx] = true;
					if (rand % 6 == 0) blockcol[blockidx] = 4;
					else if (rand % 6 == 1) blockcol[blockidx] = 3;
					else blockcol[blockidx] = 5;
				}
				else if (disttmp < 497)
				{
					tmp3 = disttmp - 375;
					if (tmp3 < 0) tmp3 *= -1;
					if(tmp3>15) blocks[blockidx] = true;

					if (rand % 10 == 0) blockcol[blockidx] = 43;
					else if (rand % 10 == 1) blockcol[blockidx] = 86;
					else if (rand % 10 == 2) blockcol[blockidx] = 129;
					else if (rand % 10 == 3) blockcol[blockidx] = 172;
					else if (rand % 10 < 7) blockcol[blockidx] = 8;
					else blockcol[blockidx] = 9;

					if(rand%300==0) blockcol[blockidx] = 149;
					if (rand % 1000000 == 0)
					{
						blockcol[blockidx] = 0;
					}
				}
				else if (disttmp < 499)
				{
					blocks[blockidx] = true;
					blockcol[blockidx] = 180;

					if (k > 957) blockcol[blockidx] = 204;
					if (k > 960) blockcol[blockidx] = 215;

					if (k < 50) blockcol[blockidx] = 204;
					if (k < 48) blockcol[blockidx] = 215;
					

					if (tmp2 > 950) blockcol[blockidx] = 24;

					if (tmp2 < 75)blockcol[blockidx] = 18;

					if (j > 950) blockcol[blockidx] = 29;

					if (j < 75)blockcol[blockidx] = 101;
				}


}


__global__ void addKernel(uint8_t* buffer, double* vecl, bool* blocks, double* norm0, double* norm1, double* norm2, double* point0, double* point1, double* point2, int* mir, double pos0, double pos1, double pos2, double vec0, double vec1, double vec2, double addy0, double addy1, double addy2, double addz0, double addz1, double addz2, int nbblocks, uint8_t* blockcol, uint8_t* stars, int nbframe, int skyr,int skyg, int skyb)
{
	int i;

	double sp1 = 40000;
	double sp2 = 0.001;


	double vecn0, vecn1, vecn2;
	double cpos0, cpos1, cpos2;
	double tpos0, tpos1, tpos2;
	int cnx, cny, cnz;

	double tmin;
	double ttmp;
	double alpha;

	double px, py, pz;

	double qa, qb, qc;
	double discr;

	double t1, t2;
	double t1f, t2f;
	double tcont;

	double skyfac;

	double kb, kc, kd;

	int col, colr, colg;

	int tmp = blockIdx.x * blockDim.x + threadIdx.x;
	int tmpx = tmp % 1280;
	int tmpy = (tmp - tmpx) / 1280;

	int coll;
	int cnx2;
	int cface;

	int rnbwv;

	double u, v;

	int blockidx;
	uint8_t currblock;

	uint8_t uv;

	vecn0 = vec0 + tmpx * addy0 + tmpy * addz0;
	vecn1 = vec1 + tmpx * addy1 + tmpy * addz1;
	vecn2 = vec2 + tmpx * addy2 + tmpy * addz2;

	vecn0 /= vecl[tmp];
	vecn1 /= vecl[tmp];
	vecn2 /= vecl[tmp];


	qa = vecn0 * vecn0 + vecn1 * vecn1 + vecn2 * vecn2;
	qb = 2 * (vecn0*pos0+ vecn1 * pos1 + vecn2 * pos2 ) - 1000 * (vecn0+vecn1+vecn2);
	qc = pos0 * pos0 + pos1 * pos1 + pos2 * pos2 - 1000 * (pos0+pos1+pos2-500);

	discr = qb * qb - 4 * qa * qc;

	if (discr <= 0)
	{
		u = 2000*((0.5 + atan2(vecn1, vecn0) / (2.0 * M_PI)));
		v =2000*((0.5 + asin(vecn2) / M_PI));
		uv = stars[(int)u + 2000 * (int)v];

		qa = vecn0 * vecn0 + vecn1 * vecn1 + vecn2 * vecn2;
		qb = 2 * (vecn0 * pos0 + vecn1 * pos1 + vecn2 * pos2) - 1000 * (vecn0 + vecn1 + vecn2);
		qc = pos0 * pos0 + pos1 * pos1 + pos2 * pos2 - 1000 * (pos0 + pos1 + pos2) - 250000;

		discr = qb * qb - 4 * qa * qc;

		if (discr <= 0) skyfac = 0;
		else
		{
			t1 = ((-1) * qb - sqrt(discr)) / (2.0 * qa);
			t2 = ((-1) * qb + sqrt(discr)) / (2.0 * qa);

			if (t1 < 0 && t2 < 0) skyfac = 0;
			else
			{
				if (t2 > t1)
				{
					t1f = t1;
					t2f = t2;
				}
				else
				{
					t1f = t2;
					t2f = t1;
				}
				if (t1f < 0) t1f = 0;

				kb = vecn0 * vecn0 + vecn1 * vecn1 + vecn2 * vecn2;
				kc = 2 * vecn0 * pos0 - 1000 * vecn0;
				kc += 2 * vecn1 * pos1 - 1000 * vecn1;
				kc += 2 * vecn2 * pos2 - 1000 * vecn2;
				kd = pos0 * pos0 - 1000 * (pos0-250);
				kd += pos1 * pos1 - 1000 * (pos1 - 250);
				kd += pos2 * pos2 - 1000 * (pos2 - 250);

				skyfac = skyfunc(-2000 * sp2, kb, kc, kd, 1000000*sp2, sp2, t2f) - skyfunc(-2000*sp2, kb, kc, kd, 1000000*sp2,sp2, t1f);
				skyfac /= sp1;
				if (skyfac > 1)skyfac = 1;

			}
		}

		buffer[4 * tmp] = skyfac*skyr + (1-skyfac)* uv;
		buffer[4 * tmp + 1] = skyfac * skyg + (1 - skyfac) * uv;
		buffer[4 * tmp + 2] = skyfac * skyb + (1 - skyfac) * uv;
		buffer[4 * tmp + 3] = 255;
	}
	else
	{
		t1 = ((-1) * qb - sqrt(discr)) / (2.0 * qa);
		t2 = ((-1) * qb + sqrt(discr)) / (2.0 * qa);

		if (t1 < 0 && t2 < 0)
		{
			u = 2000 * ((0.5 + atan2(vecn1, vecn0) / (2.0 * M_PI)));
			v = 2000 * ((0.5 + asin(vecn2) / M_PI));
			uv = stars[(int)u + 2000 * (int)v];

			qa = vecn0 * vecn0 + vecn1 * vecn1 + vecn2 * vecn2;
			qb = 2 * (vecn0 * pos0 + vecn1 * pos1 + vecn2 * pos2) - 1000 * (vecn0 + vecn1 + vecn2);
			qc = pos0 * pos0 + pos1 * pos1 + pos2 * pos2 - 1000 * (pos0 + pos1 + pos2) - 250000;

			discr = qb * qb - 4 * qa * qc;

			if (discr <= 0) skyfac = 0;
			else
			{
				t1 = ((-1) * qb - sqrt(discr)) / (2.0 * qa);
				t2 = ((-1) * qb + sqrt(discr)) / (2.0 * qa);

				if (t1 < 0 && t2 < 0) skyfac = 0;
				else
				{
					if (t2 > t1)
					{
						t1f = t1;
						t2f = t2;
					}
					else
					{
						t1f = t2;
						t2f = t1;
					}
					if (t1f < 0) t1f = 0;

					kb = vecn0 * vecn0 + vecn1 * vecn1 + vecn2 * vecn2;
					kc = 2 * vecn0 * pos0 - 1000 * vecn0;
					kc += 2 * vecn1 * pos1 - 1000 * vecn1;
					kc += 2 * vecn2 * pos2 - 1000 * vecn2;
					kd = pos0 * pos0 - 1000 * (pos0 - 250);
					kd += pos1 * pos1 - 1000 * (pos1 - 250);
					kd += pos2 * pos2 - 1000 * (pos2 - 250);

					skyfac = skyfunc(-2000 * sp2, kb, kc, kd, 1000000*sp2, sp2, t2f) - skyfunc(-2000 * sp2, kb, kc, kd, 1000000*sp2, sp2, t1f);
					skyfac /= sp1;
					if (skyfac > 1)skyfac = 1;

				}
			}

			buffer[4 * tmp] = skyfac * skyr + (1 - skyfac) * uv;
			buffer[4 * tmp + 1] = skyfac * skyg + (1 - skyfac) * uv;
			buffer[4 * tmp + 2] = skyfac * skyb + (1 - skyfac) * uv;
			buffer[4 * tmp + 3] = 255;
		}
		else
		{
			if (t1 * t2 > 0)
			{
				if (t1 < t2) tcont = t1;
				else tcont = t2;

				cpos0 = pos0 + tcont * vecn0;
				cpos1 = pos1 + tcont * vecn1;
				cpos2 = pos2 + tcont * vecn2;
			}
			else
			{
				cpos0 = pos0;
				cpos1 = pos1;
				cpos2 = pos2;
			}

			px = fmod(cpos0, 1.0);
			py = fmod(cpos1, 1.0);
			pz = fmod(cpos2, 1.0);

			if (px < 0) px++;
			if (py < 0) py++;
			if (pz < 0) pz++;

			cnx = cpos0 - px;
			cny = cpos1 - py;
			cnz = cpos2 - pz;


			if ((cnx + cny + cnz) % 2 != 0)
			{
				if (px < py)
				{
					if (px < pz)
					{
						if (px < 1 - py)
						{
							if (px < 1 - pz)
							{
								px++;
								cnx--;
							}
							else
							{
								pz--;
								cnz++;
							}
						}
						else
						{
							if (1 - py < 1 - pz)
							{
								py--;
								cny++;
							}
							else
							{
								pz--;
								cnz++;
							}
						}
					}
					else
					{
						if (pz < 1 - py)
						{
							pz++;
							cnz--;
						}
						else
						{
							py--;
							cny++;
						}
					}
				}
				else
				{
					if (py < pz)
					{
						if (py < 1 - px)
						{
							if (py < 1 - pz)
							{
								py++;
								cny--;
							}
							else
							{
								pz--;
								cnz++;
							}
						}
						else
						{
							if (1 - px < 1 - pz)
							{
								px--;
								cnx++;
							}
							else
							{
								pz--;
								cnz++;
							}
						}
					}
					else
					{
						if (pz < 1 - px)
						{
							pz++;
							cnz--;
						}
						else
						{
							px--;
							cnx++;
						}
					}
				}
			}

			tpos0 = cpos0;
			tpos1 = cpos1;
			tpos2 = cpos2;

			cpos0 = px;
			cpos1 = py;
			cpos2 = pz;

			tmin = 4;


			for (i = 0; i < 12; i++)
			{
				ttmp = (point0[i] - cpos0) * norm0[i] + (point1[i] - cpos1) * norm1[i] + (point2[i] - cpos2) * norm2[i];
				ttmp /= norm0[i] * vecn0 + norm1[i] * vecn1 + norm2[i] * vecn2;

				if (ttmp > 0 && ttmp < tmin)
				{
					tmin = ttmp;
					coll = i;
				}
			}

			cnx -= norm0[coll];
			cny -= norm1[coll];
			cnz -= norm2[coll];

			cpos0 += tmin * vecn0;
			cpos1 += tmin * vecn1;
			cpos2 += tmin * vecn2;

			tpos0 += tmin * vecn0;
			tpos1 += tmin * vecn1;
		    tpos2 += tmin * vecn2;

			cpos0 += norm0[coll];
			cpos1 += norm1[coll];
			cpos2 += norm2[coll];

			cface = mir[coll];

			if (cnx < 0 || cnx >= nbblocks || cny < 0 || cny >= nbblocks || cnz < 0 || cnz >= nbblocks)
			{
				currblock = 0;
			}
			else
			{
				if (cnx % 2 != 0) cnx2 = (cnx - 1) / 2;
				else cnx2 = cnx / 2;

				blockidx = cnx2 + nbblocks * cny + nbblocks * nbblocks * cnz;

				currblock = blocks[blockidx];
			}

			while (!currblock && sqrt((tpos0-500.0)*(tpos0-500.0)+(tpos1-500.0)*(tpos1-500.0)+(tpos2-500.0)*(tpos2-500.0)) < 500.0)
			{
				tmin = 4.0;


				for (i = 0; i < 12; i++)
				{
					if (i != cface) {
						ttmp = (point0[i] - cpos0) * norm0[i] + (point1[i] - cpos1) * norm1[i] + (point2[i] - cpos2) * norm2[i];
						ttmp /= norm0[i] * vecn0 + norm1[i] * vecn1 + norm2[i] * vecn2;

						if (ttmp > 0 && ttmp < tmin)
						{
							tmin = ttmp;
							coll = i;
						}
					}
				}

				cnx -= norm0[coll];
				cny -= norm1[coll];
				cnz -= norm2[coll];

				cpos0 += tmin * vecn0;
				cpos1 += tmin * vecn1;
				cpos2 += tmin * vecn2;

				tpos0 += tmin * vecn0;
				tpos1 += tmin * vecn1;
				tpos2 += tmin * vecn2;

				cpos0 += norm0[coll];
				cpos1 += norm1[coll];
				cpos2 += norm2[coll];

				cface = mir[coll];

				if (cnx < 0 || cnx >= nbblocks || cny < 0 || cny >= nbblocks || cnz < 0 || cnz >= nbblocks)
				{
					currblock = false;
				}
				else
				{
					if (cnx % 2 != 0) cnx2 = (cnx - 1) / 2;
					else cnx2 = cnx / 2;
					cnx2 = cnx/2;

					blockidx = cnx2 + nbblocks * cny + nbblocks * nbblocks * cnz;

					currblock = blocks[blockidx];
				}
			}

			
			if (currblock)
			{
				col = blockcol[blockidx];
				if (col == 255)
				{
					rnbwv = rnbw(nbframe);

					colr = rnbwv % 256;
					rnbwv -= colr;
					rnbwv /= 256;
					colg = rnbwv % 256;
					rnbwv -= colg;
					rnbwv /= 256;
					col = rnbwv % 256;

					alpha = 1 - 0.042 * cface;
					buffer[4 * tmp] = colr * alpha;
					buffer[4 * tmp + 1] = colg * alpha;
					buffer[4 * tmp + 2] = col * alpha;
					buffer[4 * tmp + 3] = 255;
				}
				else
				{
					colr = col % 6;
					col -= colr;
					col /= 6;
					colg = col % 6;
					col -= colg;
					col /= 6;
					col %= 6;

					colr *= (255 / 5);
					colg *= (255 / 5);
					col *= (255 / 5);

					alpha = 1 - 0.042 * cface;
					buffer[4 * tmp] = colr * alpha;
					buffer[4 * tmp + 1] = colg * alpha;
					buffer[4 * tmp + 2] = col * alpha;
					buffer[4 * tmp + 3] = 255;
				}
			}
			else
			{
				buffer[4 * tmp] = skyr;
				buffer[4 * tmp + 1] = skyg;
				buffer[4 * tmp + 2] = skyb;
				buffer[4 * tmp + 3] = 255;
			}
		}
	}
}


void cudaInit(bool* blockstmp)
{
	double dist = 1;
	double sqsz = 0.01 / 4;
	int tmpx, tmpy;

	double* vecltmp = new double[1280 * 720];

	double vec0, vec1, vec2;
	double addy0, addy1, addy2;
	double addz0, addz1, addz2;
	double vecn0, vecn1, vecn2;
	double x00 = 1, x01 = 0, x02 = 0;
	double x10 = 0, x11 = 1, x12 = 0;
	double x20 = 0, x21 = 0, x22 = 1;
	double multy = (1 - 1280) * sqsz / 2;
	double multz = (720 - 1) * sqsz / 2;

	double* norm0tmp = new double[12];
	double* norm1tmp = new double[12];
	double* norm2tmp = new double[12];
	double* point0tmp = new double[12];
	double* point1tmp = new double[12];
	double* point2tmp = new double[12];

	int* mirtmp = new int[12];

	uint8_t* blockcoltmp = new uint8_t[nbblocks * nbblocks * nbblocks];
	uint8_t* starstmp = new uint8_t[2000 * 2000];


	hipSetDevice(0);
	hipMalloc((void**)&buffer, 4 * 1280 * 720 * sizeof(uint8_t));
	hipMalloc((void**)&vecl, 1280 * 720 * sizeof(double));
	hipMalloc((void**)&blocks, nbblocks * nbblocks * nbblocks * sizeof(bool));

	hipMalloc((void**)&norm0, 12 * sizeof(double));
	hipMalloc((void**)&norm1, 12 * sizeof(double));
	hipMalloc((void**)&norm2, 12 * sizeof(double));

	hipMalloc((void**)&point0, 12 * sizeof(double));
	hipMalloc((void**)&point1, 12 * sizeof(double));
	hipMalloc((void**)&point2, 12 * sizeof(double));

	hipMalloc((void**)&mir, 12 * sizeof(int));

	hipMalloc((void**)&blockcol, nbblocks*nbblocks*nbblocks * sizeof(uint8_t));

	hipMalloc((void**)&stars, 2000*2000* sizeof(uint8_t));

	vec0 = dist * x00 + multy * x10 + multz * x20;
	vec1 = dist * x01 + multy * x11 + multz * x21;
	vec2 = dist * x02 + multy * x12 + multz * x22;

	addy0 = sqsz * x10;
	addy1 = sqsz * x11;
	addy2 = sqsz * x12;

	addz0 = -sqsz * x20;
	addz1 = -sqsz * x21;
	addz2 = -sqsz * x22;

	for (int i = 0; i < 1280 * 720; i++)
	{
		tmpx = i % 1280;
		tmpy = (i - tmpx) / 1280;

		vecn0 = vec0 + tmpx * addy0 + tmpy * addz0;
		vecn1 = vec1 + tmpx * addy1 + tmpy * addz1;
		vecn2 = vec2 + tmpx * addy2 + tmpy * addz2;

		vecltmp[i] = sqrt(vecn0 * vecn0 + vecn1 * vecn1 + vecn2 * vecn2);
	}



	point0tmp[0] = 0.5;
	point1tmp[0] = 0.5;
	point2tmp[0] = 1.5;

	point0tmp[1] = 0.5;
	point1tmp[1] = 0.5;
	point2tmp[1] = 1.5;

	point0tmp[2] = 0.5;
	point1tmp[2] = 0.5;
	point2tmp[2] = 1.5;

	point0tmp[3] = 0.5;
	point1tmp[3] = 0.5;
	point2tmp[3] = 1.5;

	point0tmp[4] = 1.5;
	point1tmp[4] = 0.5;
	point2tmp[4] = 0.5;

	point0tmp[5] = 0.5;
	point1tmp[5] = 1.5;
	point2tmp[5] = 0.5;

	point0tmp[6] = -0.5;
	point1tmp[6] = 0.5;
	point2tmp[6] = 0.5;

	point0tmp[7] = 0.5;
	point1tmp[7] = -0.5;
	point2tmp[7] = 0.5;

	point0tmp[8] = 0.5;
	point1tmp[8] = 0.5;
	point2tmp[8] = -0.5;

	point0tmp[9] = 0.5;
	point1tmp[9] = 0.5;
	point2tmp[9] = -0.5;

	point0tmp[10] = 0.5;
	point1tmp[10] = 0.5;
	point2tmp[10] = -0.5;

	point0tmp[11] = 0.5;
	point1tmp[11] = 0.5;
	point2tmp[11] = -0.5;

	norm0tmp[0] = -1;
	norm1tmp[0] = 0;
	norm2tmp[0] = -1;

	norm0tmp[1] = 0;
	norm1tmp[1] = -1;
	norm2tmp[1] = -1;

	norm0tmp[2] = 1;
	norm1tmp[2] = 0;
	norm2tmp[2] = -1;

	norm0tmp[3] = 0;
	norm1tmp[3] = 1;
	norm2tmp[3] = -1;

	norm0tmp[4] = -1;
	norm1tmp[4] = -1;
	norm2tmp[4] = 0;

	norm0tmp[5] = 1;
	norm1tmp[5] = -1;
	norm2tmp[5] = 0;

	norm0tmp[6] = 1;
	norm1tmp[6] = 1;
	norm2tmp[6] = 0;

	norm0tmp[7] = -1;
	norm1tmp[7] = 1;
	norm2tmp[7] = 0;

	norm0tmp[8] = -1;
	norm1tmp[8] = 0;
	norm2tmp[8] = 1;

	norm0tmp[9] = 0;
	norm1tmp[9] = -1;
	norm2tmp[9] = 1;

	norm0tmp[10] = 1;
	norm1tmp[10] = 0;
	norm2tmp[10] = 1;

	norm0tmp[11] = 0;
	norm1tmp[11] = 1;
	norm2tmp[11] = 1;

	mirtmp[0] = 10;
	mirtmp[1] = 11;
	mirtmp[2] = 8;
	mirtmp[3] = 9;
	mirtmp[4] = 6;
	mirtmp[5] = 7;
	mirtmp[6] = 4;
	mirtmp[7] = 5;
	mirtmp[8] = 2;
	mirtmp[9] = 3;
	mirtmp[10] = 0;
	mirtmp[11] = 1;

	hipMemcpy(vecl, vecltmp, 1280 * 720 * sizeof(double), hipMemcpyHostToDevice);
	
	hipMemcpy(norm0, norm0tmp, 12 * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(norm1, norm1tmp, 12 * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(norm2, norm2tmp, 12 * sizeof(double), hipMemcpyHostToDevice);

	hipMemcpy(point0, point0tmp, 12 * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(point1, point1tmp, 12 * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(point2, point2tmp, 12 * sizeof(double), hipMemcpyHostToDevice);


	hipMemcpy(mir, mirtmp, 12 * sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy(stars, starstmp,2000*2000 * sizeof(uint8_t), hipMemcpyHostToDevice);

	setstars << <2000 * 2000 / 500, 500 >> > (stars);
	hipDeviceSynchronize();

	setplanet << <nbblocks*nbblocks*nbblocks / 500, 500 >> > (blocks,blockcol);
	hipDeviceSynchronize();

	hipMemcpy(blockstmp, blocks, nbblocks * nbblocks * nbblocks * sizeof(bool), hipMemcpyDeviceToHost);

}

void cudaExit()
{
	hipFree(buffer);
	hipFree(vecl);
	hipDeviceReset();
}

void cudathingy(uint8_t* pixels, double pos0, double pos1, double pos2, double vec0, double vec1, double vec2, double addy0, double addy1, double addy2, double addz0, double addz1, double addz2, int remidx, int addidx, int buildidx, uint8_t col, int nbframe)
{
	if (remidx != -1)
	{
		remblock << <1, 1 >> > (blocks,remidx);
		hipDeviceSynchronize();
	}
	if (addidx != -1)
	{
		addblock << <1, 1 >> > (blocks, addidx);
		hipDeviceSynchronize();
		
	}
	if (buildidx != -1)
	{
		changecol << <1, 1 >> > (blockcol, buildidx,col);
		hipDeviceSynchronize();
	}

	if (sqrt((pos0 - 500) * (pos0 - 500) + (pos1 - 500) * (pos1 - 500) + (pos2 - 500) * (pos2 - 500)) < 32)
	{
		skyr = 255;
		skyg = 174;
		skyb = 201;
	}

	addKernel <<<(int)(1280 * 720 / 600), 600 >>> (buffer, vecl, blocks, norm0, norm1, norm2, point0, point1, point2, mir, pos0, pos1, pos2, vec0, vec1, vec2, addy0, addy1, addy2, addz0, addz1, addz2, nbblocks, blockcol,stars,nbframe,skyr,skyg,skyb);
	hipDeviceSynchronize();
	hipMemcpy(pixels, buffer, 4 * 1280 * 720 * sizeof(uint8_t), hipMemcpyDeviceToHost);
}